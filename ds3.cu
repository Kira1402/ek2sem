
#include <hip/hip_runtime.h>
#include <iostream> 
#define BLOCK_SIZE 32
using namespace std;

__global__ void matmul_naive_1(double *a, double *b, double *c, int matrix_dim) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < matrix_dim && y < matrix_dim) {
        float tmp = 0.0;
        for (int i = 0; i < matrix_dim; ++i) {
            tmp += a[x * matrix_dim + i] * b[i * matrix_dim + y];
        }
        c[x * matrix_dim + y] = tmp;
    }
}

__global__ void matmul_naive_2(double *a, double *b, double *c, int matrix_dim) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < matrix_dim && y < matrix_dim) {
        float tmp = 0.0;
        for (int i = 0; i < matrix_dim; ++i) {
            tmp += a[y * matrix_dim + i] * b[i * matrix_dim + x];
        }
        c[y * matrix_dim + x] = tmp;
    }
}

int check_if_equal(double *a, double *b, int N){

    for (int i = 0; i != N; i++){
        if (a[i] != b[i]){
            return 1;
        }
    }
    
    return 0;
}

void random_fill(double *array, int N, double random_lowest, double random_highest){

    const long max_rand = 1000000L;
    static double timep = 0.0;
    timep += 1.0;
    srandom(time(NULL) + timep);
    for (int i = 0; i != N; ++i){
    array[i] = random_lowest+(random_highest - random_lowest)*(random() % max_rand)/max_rand;
    }
}


void matmul_simple_using_host(double *a, double *b, double *c, int N){


}

int main() {

    double *a, *b, *c_host, *c_device; 
    double *d_a, *d_b, *d_c;
    const int N = 12; 
    const int NN = pow(N, 2);
    double random_lowest = 1.0; 
    double random_highest = 10.0; 
    int size = NN * sizeof(double);

    dim3 gridDim(N / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1, 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Alloc space for host copies of a, b, c and setup input values
    a        = (double *)malloc(size);
    b        = (double *)malloc(size);
    c_host   = (double *)malloc(size);
    c_device = (double *)malloc(size);

    random_fill(a, NN, random_lowest, random_highest);
    random_fill(b, NN, random_lowest, random_highest);

    for (int x = 0; x != N; x++){
        for (int y = 0; y != N; y++){
            float tmp = 0.0;
            for (int i = 0; i != N; ++i) {
                tmp += a[x * N + i] * b[i * N + y];
            }
            c_host[x * N + y] = tmp;
        }
    }

    // Alloc space for device copies of a, b
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    matmul_naive_1<<<gridDim,blockDim>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c_device, d_c, size, hipMemcpyDeviceToHost);

    //check if two results are equal
    if (check_if_equal(c_device, c_host, NN)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }


    matmul_naive_2<<<gridDim,blockDim>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c_device, d_c, size, hipMemcpyDeviceToHost);

    //check if two results are equal
    if (check_if_equal(c_device, c_host, NN)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }

    hipFree(d_a); hipFree(d_b); hipFree(d_c); 
    free(a); free(b); free(c_host); free(c_device);
    return 0;
}
